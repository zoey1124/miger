
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_A(double* A, int N, int M)
{
    double d = 0.0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {

#pragma unroll(100)
        for (int j = 0; j < M; ++j) {
            d += A[idx];
        }

        A[idx] = d;

    }
}

__global__ void kernel_B(double* A, int N, int M)
{
    double d = 0.0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {

#pragma unroll(100)
        for (int j = 0; j < M; ++j) {
            d += A[idx];
        }

        A[idx] = d;

    }
}

__global__ void kernel_C(double* A, const double* B, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Strided memory access: warp 0 accesses (0, stride, 2*stride, ...), warp 1 accesses
    // (1, stride + 1, 2*stride + 1, ...).
    const int stride = 16;
    int strided_idx = threadIdx.x * stride + blockIdx.x % stride + (blockIdx.x / stride) * stride * blockDim.x;

    if (strided_idx < N) {
        A[idx] = B[strided_idx] + B[strided_idx];
    }
}

int main() {

    double* A;
    double* B;

    int N = 80 * 2048 * 100; // 100 * maximum number of resident threads on V100
    size_t sz = N * sizeof(double);

    hipMalloc((void**) &A, sz);
    hipMalloc((void**) &B, sz);

    hipMemset(A, 0, sz);
    hipMemset(B, 0, sz);

    int threadsPerBlock = 64;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int M = 10000;

    kernel_A<<<numBlocks, threadsPerBlock>>>(A, N, M);

    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_B), hipFuncAttributeMaxDynamicSharedMemorySize, 96 * 1024);
    kernel_B<<<numBlocks, threadsPerBlock, 96 * 1024>>>(A, N, M);

//    kernel_C<<<numBlocks, threadsPerBlock>>>(A, B, N);

    hipDeviceSynchronize();

}
